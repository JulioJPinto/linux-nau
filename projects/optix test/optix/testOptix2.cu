#include "hip/hip_runtime.h"

#include <optix.h>
#include "LaunchParams2.h" // our launch params
#include <vec_math.h> // NVIDIAs math utils


extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}
//  a single ray type
enum { SURFACE_RAY_TYPE=0, RAY_TYPE_COUNT };

// pack and unpack payload pointer from
// Ingo Wald Optix 7 course
// https://gitlab.com/ingowald/optix7course

static __forceinline__ __device__
void *unpackPointer( uint32_t i0, uint32_t i1 ) {
    const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr ); 
    return ptr;
}

static __forceinline__ __device__
void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 ) {
    const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T *getPRD() { 
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
}

// -------------------------------------------------------
// closest hit computes color based lolely on the triangle normal

extern "C" __global__ void __closesthit__radiance()
{
    float3 &prd = *(float3*)getPRD<float3>();

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  
    // compute triangle normal:
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {  
        // get barycentric coordinates
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;
        // compute pixel texture coordinate
        const float4 tc
          = (1.f-u-v) * sbtData.vertexD.texCoord0[index.x]
          +         u * sbtData.vertexD.texCoord0[index.y]
          +         v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);
        prd= make_float3(fromTexture);
    }
    else
        prd = sbtData.color;
}


  

// nothing to do in here
extern "C" __global__ void __anyhit__radiance() {
}


// miss sets the bacgground color
extern "C" __global__ void __miss__radiance() {

    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(0.0f, 0.0f, 1.0f);
}


extern "C" __global__ void __raygen__renderFrame() {

    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  
    
    // ray payload
    float3 pixelColorPRD = make_float3(1.f);
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );  

    // compute ray direction
    // normalized screen plane position, in [-1, 1]^2
    const float2 screen(make_float2(ix+.5f,iy+.5f)
                    / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);
  
    // note: nau already takes into account the field of view and ratio when computing 
    // camera horizontal and vertival
    float3 rayDir = normalize(camera.direction
                           + screen.x  * camera.horizontal
                           + screen.y * camera.vertical);
    
    // trace primary ray
    optixTrace(optixLaunchParams.traversable,
             camera.position,
             rayDir,
             0.f,    // tmin
             1e20f,  // tmax
             0.0f,   // rayTime
             OptixVisibilityMask( 255 ),
             OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
             SURFACE_RAY_TYPE,             // SBT offset
             RAY_TYPE_COUNT,               // SBT stride
             SURFACE_RAY_TYPE,             // missSBTIndex 
             u0, u1 );

    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*pixelColorPRD.x);
    const int g = int(255.0f*pixelColorPRD.y);
    const int b = int(255.0f*pixelColorPRD.z);

    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000
      | (r<<0) | (g<<8) | (b<<16);
    // compute index
    const uint32_t fbIndex = ix + iy*optixGetLaunchDimensions().x;
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}
  

